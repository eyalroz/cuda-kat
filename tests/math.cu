#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "common.cuh"
#include <kat/on_device/math.cuh>

namespace kernels {

template <typename I>
__global__ void try_out_integral_math_functions(I* results, I* __restrict expected)
{
	size_t i { 0 };
	bool print_first_indices_for_each_function { false };

	auto maybe_print = [&](const char* section_title) {
		if (print_first_indices_for_each_function) {
			printf("%-30s tests start at index  %3d\n", section_title, (int) i);
		}
	};

	results[i] = kat::strictly_between<I>( I{   0 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   1 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   4 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   5 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   6 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{   8 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{   9 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{  10 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{  11 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{ 123 }, I{  5 }, I{  10 } ); expected[i++] = false;

	maybe_print("between_or_equal");
	results[i] = kat::between_or_equal<I>( I{   1 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{   4 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{   5 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   6 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   8 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   9 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{  10 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{  11 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{ 123 }, I{  5 }, I{  10 } ); expected[i++] = false;

	maybe_print("is_power_of_2");
	results[i] = kat::is_power_of_2<I>(I{ 1}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 2}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 4}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 7}); expected[i++] = false;
	results[i] = kat::is_power_of_2<I>(I{32}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{33}); expected[i++] = false;

	maybe_print("modular_increment");
	results[i] = kat::modular_increment<I>(I{ 0}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 1}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 0}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_increment<I>(I{ 1}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_increment<I>(I{ 2}, I{ 3}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 3}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_increment<I>(I{ 4}, I{ 3}); expected[i++] = I{ 2 };

	maybe_print("modular_decrement");
	results[i] = kat::modular_decrement<I>(I{ 0}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 1}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 0}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_decrement<I>(I{ 1}, I{ 3}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 2}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_decrement<I>(I{ 3}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_decrement<I>(I{ 4}, I{ 3}); expected[i++] = I{ 0 };

	maybe_print("ipow");
	results[i] = kat::ipow<I>(I{ 0 },   1 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 0 },   2 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 0 }, 100 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 1 },   0 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 },   1 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 },   2 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 }, 100 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 3 },   0 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 3 },   1 ); expected[i++] = I{  3 };
	results[i] = kat::ipow<I>(I{ 3 },   2 ); expected[i++] = I{  9 };
	results[i] = kat::ipow<I>(I{ 3 },   4 ); expected[i++] = I{ 81 };

	maybe_print("unsafe div_rounding_up");
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };

	maybe_print("div_rounding_up");
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( std::numeric_limits<I>::max()    , std::numeric_limits<I>::max() - 1 ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( std::numeric_limits<I>::max() - 1, std::numeric_limits<I>::max()     ); expected[i++] = I{   1 };

	maybe_print("round_down");
	results[i] = kat::round_down<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{   1 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{ 123 };
	results[i] = kat::round_down<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{ 123 };

	maybe_print("round_down_to_full_warps");
	results[i] = kat::round_down_to_full_warps<I>( I{   0 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{   1 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{   8 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  16 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  31 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  32 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_down_to_full_warps<I>( I{  33 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_down_to_full_warps<I>( I{ 125 } ); expected[i++] = I{ 96 };

	// TODO: Consider testing rounding-up with negative dividends

	maybe_print("unsafe round_up");
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::round_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up<I>( I{  63 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::unsafe::round_up<I>( I{  64 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::unsafe::round_up<I>( I{  65 }, I{  32 } ); expected[i++] = I{  96 };

	maybe_print("round_up");
	results[i] = kat::round_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up<I>( I{  63 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::round_up<I>( I{  64 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::round_up<I>( I{  65 }, I{  32 } ); expected[i++] = I{  96 };
	results[i] = kat::round_up<I>( std::numeric_limits<I>::max() - 1, std::numeric_limits<I>::max() ); expected[i++] = I{ std::numeric_limits<I>::max() };

	maybe_print("round_down_to_power_of_2");
	results[i] = kat::round_down_to_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   3 }, I{   1 } ); expected[i++] = I{   3 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   4 }, I{   1 } ); expected[i++] = I{   4 };
	results[i] = kat::round_down_to_power_of_2<I>( I{ 123 }, I{   1 } ); expected[i++] = I{ 123 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   3 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   4 }, I{   2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_down_to_power_of_2<I>( I{ 123 }, I{   2 } ); expected[i++] = I{ 122 };

	maybe_print("round_up_to_power_of_2");
	results[i] = kat::round_up_to_power_of_2<I>( I{  1 }, I{  1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  2 }, I{  1 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  3 }, I{  1 } ); expected[i++] = I{   3 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  4 }, I{  1 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{ 23 }, I{  1 } ); expected[i++] = I{  23 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  1 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  2 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  3 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  4 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{ 63 }, I{  2 } ); expected[i++] = I{  64 };

	maybe_print("unsafe round_up_to_power_of_2");
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  1 }, I{  1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  2 }, I{  1 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  3 }, I{  1 } ); expected[i++] = I{   3 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  4 }, I{  1 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{ 23 }, I{  1 } ); expected[i++] = I{  23 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  1 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  2 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  3 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  4 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{ 63 }, I{  2 } ); expected[i++] = I{  64 };

	maybe_print("round_up_to_full_warps");
	results[i] = kat::round_up_to_full_warps<I>( I{   0 } ); expected[i++] = I{  0 };
	results[i] = kat::round_up_to_full_warps<I>( I{   1 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{   8 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  16 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  31 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  32 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  33 } ); expected[i++] = I{ 64 };
	results[i] = kat::round_up_to_full_warps<I>( I{  63 } ); expected[i++] = I{ 64 };

	maybe_print("gcd");
	results[i] = kat::gcd<I>( I{   1 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   2 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   1 }, I{   2 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   2 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::gcd<I>( I{   8 }, I{   4 } ); expected[i++] = I{  4 };
	results[i] = kat::gcd<I>( I{   4 }, I{   8 } ); expected[i++] = I{  4 };
	results[i] = kat::gcd<I>( I{  10 }, I{   6 } ); expected[i++] = I{  2 };
	results[i] = kat::gcd<I>( I{ 120 }, I{  70 } ); expected[i++] = I{ 10 };
	results[i] = kat::gcd<I>( I{  70 }, I{ 120 } ); expected[i++] = I{ 10 };
	results[i] = kat::gcd<I>( I{  97 }, I{ 120 } ); expected[i++] = I{  1 };

	maybe_print("lcm");
	results[i] = kat::lcm<I>( I{   1 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::lcm<I>( I{   2 }, I{   1 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   1 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   2 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   5 }, I{   3 } ); expected[i++] = I{ 15 };
	results[i] = kat::lcm<I>( I{   8 }, I{   4 } ); expected[i++] = I{  8 };
	results[i] = kat::lcm<I>( I{   4 }, I{   8 } ); expected[i++] = I{  8 };
	results[i] = kat::lcm<I>( I{  10 }, I{   6 } ); expected[i++] = I{ 30 };

	maybe_print("is_even");
	results[i] = kat::is_even<I>( I{   0 } ); expected[i++] = true;
	results[i] = kat::is_even<I>( I{   1 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{   2 } ); expected[i++] = true;
	results[i] = kat::is_even<I>( I{   3 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{ 123 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{ 124 } ); expected[i++] = true;

	maybe_print("is_odd");
	results[i] = kat::is_odd<I>( I{   0 } ); expected[i++] = false;
	results[i] = kat::is_odd<I>( I{   1 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{   2 } ); expected[i++] = false;
	results[i] = kat::is_odd<I>( I{   3 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{ 123 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{ 124 } ); expected[i++] = false;

	maybe_print("log2");
	results[i] = kat::log2<I>( I{   1 } ); expected[i++] = 0;
	results[i] = kat::log2<I>( I{   2 } ); expected[i++] = 1;
	results[i] = kat::log2<I>( I{   3 } ); expected[i++] = 1;
	results[i] = kat::log2<I>( I{   4 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   6 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   7 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   8 } ); expected[i++] = 3;
	results[i] = kat::log2<I>( I{ 127 } ); expected[i++] = 6;

//	We don't have a goot integer sqrt() implementation to offer here. Perhaps
//	we could offer something based on casting to float?
//
//	results[i] = kat::sqrt<I>( I{   0 } ); expected[i++] =  0;
//	results[i] = kat::sqrt<I>( I{   1 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   2 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   3 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   4 } ); expected[i++] =  2;
//	results[i] = kat::sqrt<I>( I{   5 } ); expected[i++] =  2;
//	results[i] = kat::sqrt<I>( I{   9 } ); expected[i++] =  3;
//	results[i] = kat::sqrt<I>( I{  10 } ); expected[i++] =  3;
//	results[i] = kat::sqrt<I>( I{ 127 } ); expected[i++] = 11;

	maybe_print("div_by_power_of_2");
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I {  1 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I {  1 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I {  1 }); expected[i++] = I{ 111 };
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I {  2 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I {  2 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   2 }, I {  2 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{   3 }, I {  2 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{   4 }, I {  2 }); expected[i++] = I{   2 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I {  2 }); expected[i++] = I{  55 };
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{  15 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{  16 }, I { 16 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{  17 }, I { 16 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{  32 }, I { 16 }); expected[i++] = I{   2 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I { 16 }); expected[i++] = I{   6 };

	maybe_print("divides");
	results[i] = kat::divides<I>( I{   1 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   3 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   3 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   1 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   3 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   4 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   6 }, I{   9 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   9 }, I{   6 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   4 }, I{  24 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{  24 }, I{   4 } ); expected[i++] = false;

	maybe_print("is_divisible_by");
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   3 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   3 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   3 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   4 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   9 }, I{   6 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   6 }, I{   9 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{  24 }, I{   4 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   4 }, I{  24 } ); expected[i++] = false;

	maybe_print("is_divisible_by_power_of_2");
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   0 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   0 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   4 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  24 }, I{   4 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  72 }, I{  16 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  64 }, I{  16 } ); expected[i++] = true;

	maybe_print("power_of_2_divides");
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   4 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{   4 }, I{  24 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{  16 }, I{  72 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{  16 }, I{  64 } ); expected[i++] = true;

	maybe_print("log2_of_power_of_2");
	results[i] = kat::log2_of_power_of_2<I>( I{  1 } ); expected[i++] = I{ 0 };
	results[i] = kat::log2_of_power_of_2<I>( I{  2 } ); expected[i++] = I{ 1 };
	results[i] = kat::log2_of_power_of_2<I>( I{  4 } ); expected[i++] = I{ 2 };
	results[i] = kat::log2_of_power_of_2<I>( I{  8 } ); expected[i++] = I{ 3 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 16 } ); expected[i++] = I{ 4 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 32 } ); expected[i++] = I{ 5 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 64 } ); expected[i++] = I{ 6 };

	maybe_print("modulo_power_of_2");
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   4 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   4 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   4 } ); expected[i++] = I{ 2 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   4 } ); expected[i++] = I{ 3 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   4 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   4 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   4 } ); expected[i++] = I{ 3 };

#define NUM_INTEGER_FUNCTION_TEST_LINES 268

}

template <typename F>
__global__ void try_out_floating_point_math_functions(F* results, F* __restrict expected)
{
	size_t i { 0 };
	bool print_first_indices_for_each_function { false };

	auto maybe_print = [&](const char* section_title) {
		if (print_first_indices_for_each_function) {
			printf("%-30s tests start at index  %3d\n", section_title, (int) i);
		}
	};

	maybe_print("log2");
	results[i] = kat::log2<F>( F{    1       } ); expected[i++] =  0;
	results[i] = kat::log2<F>( F{    2       } ); expected[i++] =  1;
	results[i] = kat::log2<F>( F{    3       } ); expected[i++] =  log2(3);
	results[i] = kat::log2<F>( F{    4       } ); expected[i++] =  2;
	results[i] = kat::log2<F>( F{    6       } ); expected[i++] =  log2(6);
	results[i] = kat::log2<F>( F{    7       } ); expected[i++] =  log2(7);
	results[i] = kat::log2<F>( F{    8       } ); expected[i++] =  log2(8);
	results[i] = kat::log2<F>( F{  127       } ); expected[i++] =  log2(127);
	results[i] = kat::log2<F>( F{    0.5     } ); expected[i++] = -1;
	results[i] = kat::log2<F>( F{    0.25    } ); expected[i++] = -2;
	results[i] = kat::log2<F>( F{    0.125   } ); expected[i++] = -3;
	results[i] = kat::log2<F>( F{    0.0625  } ); expected[i++] = -4;
	results[i] = kat::log2<F>( F{    0.03125 } ); expected[i++] = -5;


#define NUM_FLOATING_POINT_FUNCTION_TEST_LINES 13

}


} // namespace kernels

template <typename T>
const auto make_exact_comparison { optional<T>{} };

namespace detail {

template <typename T>
auto tolerance_gadget(std::true_type, T x, optional<T> tolerance) {
	auto eps = tolerance.value_or(0);
	return doctest::Approx(x).epsilon(eps);
}


template <typename T>
T tolerance_gadget(std::false_type, T x, optional<T>) { return x; }

template <typename T>
std::size_t required_width_to_fit(T max)
{
//	assert(std::is_integral<I>::value, "Only integer types supported for now");
	std::stringstream ss;
	ss << std::dec << max;
	return ss.str().length();
}

} // namespace detail

template <typename T>
auto tolerance_gadget(T x, optional<T> tolerance)
{
	constexpr const auto is_arithmetic = std::is_arithmetic< std::decay_t<T> >::value;
	return
		detail::tolerance_gadget(std::integral_constant<bool, is_arithmetic>{}, x, tolerance);
}

template <typename T, typename F, typename... Is>
void check_results(
	std::string               title,
	size_t                    num_values_to_check,
	const T*  __restrict__    actual_values,
	F                         expected_value_retriever,
	optional<T>               comparison_tolerance_fraction,
	const Is* __restrict__... inputs)
{
	std::stringstream ss;
	auto index_width = detail::required_width_to_fit(num_values_to_check);

	// TODO: Consider using the maximum/minimum result values to set field widths.

	for(size_t i = 0; i < num_values_to_check; i++) {
		ss.str("");
		ss
			<< "Assertion " << std::setw(index_width) << (i+1) << " for " << title
			// << " :\n"
			<< "(" << std::make_tuple(inputs[i]...) << ")"
		;
		std::string mismatch_message { ss.str() };
		if (comparison_tolerance_fraction) {
			const auto& actual = actual_values[i];
			const auto expected = tolerance_gadget(expected_value_retriever(i), comparison_tolerance_fraction);
			CHECK_MESSAGE(actual == expected, mismatch_message);
		}
		else {
			const auto& ev = expected_value_retriever(i);
			const auto& actual = actual_values[i];
			const auto expected = expected_value_retriever(i);
			CHECK_MESSAGE(actual == expected, mismatch_message);
		}
	}
}

template <typename T, typename F, typename... Is>
void check_results(
	size_t                    num_values_to_check,
	const T*  __restrict__    actual_values,
	F                         expected_value_retriever,
	optional<T>               comparison_tolerance_fraction,
	const Is* __restrict__... inputs)
{
	return check_results(
		std::string("testcase ") + doctest::current_test_name(),
		num_values_to_check,
		actual_values,
		expected_value_retriever,
		comparison_tolerance_fraction,
		inputs...);
}

TEST_SUITE("math") {

TEST_CASE_TEMPLATE("run-time on-device integral math", I, INTEGER_TYPES)
{
	cuda::device_t device { cuda::device::current::get() };
	auto block_size { 1 };
	auto num_grid_blocks { 1 };
	auto launch_config { cuda::make_launch_config(block_size, num_grid_blocks) };
	auto device_side_results { cuda::memory::device::make_unique<I[]>(device, NUM_INTEGER_FUNCTION_TEST_LINES) };
	auto device_side_expected_results { cuda::memory::device::make_unique<I[]>(device, NUM_INTEGER_FUNCTION_TEST_LINES) };
	auto host_side_results { std::unique_ptr<I[]>(new I[NUM_INTEGER_FUNCTION_TEST_LINES]) };
	auto host_side_expected_results { std::unique_ptr<I[]>(new I[NUM_INTEGER_FUNCTION_TEST_LINES]) };

	cuda::launch(
		kernels::try_out_integral_math_functions<I>,
		launch_config,
		device_side_results.get(), device_side_expected_results.get());

	cuda::memory::copy(host_side_results.get(), device_side_results.get(), sizeof(I) * NUM_INTEGER_FUNCTION_TEST_LINES);
	cuda::memory::copy(host_side_expected_results.get(), device_side_expected_results.get(), sizeof(I) * NUM_INTEGER_FUNCTION_TEST_LINES);

	check_results(
		NUM_FLOATING_POINT_FUNCTION_TEST_LINES,
		host_side_results.get(),
		[ expected_results = host_side_expected_results.get() ](std::size_t i) { return expected_results[i]; },
		make_exact_comparison<I>
	);
}

TEST_CASE_TEMPLATE("run-time on-device floating-point math", F, FLOAT_TYPES)
{
	cuda::device_t device { cuda::device::current::get() };
	auto block_size { 1 };
	auto num_grid_blocks { 1 };
	auto launch_config { cuda::make_launch_config(block_size, num_grid_blocks) };
	auto device_side_results { cuda::memory::device::make_unique<F[]>(device, NUM_FLOATING_POINT_FUNCTION_TEST_LINES) };
	auto device_side_expected_results { cuda::memory::device::make_unique<F[]>(device, NUM_FLOATING_POINT_FUNCTION_TEST_LINES) };
	auto host_side_results { std::unique_ptr<F[]>(new F[NUM_FLOATING_POINT_FUNCTION_TEST_LINES]) };
	auto host_side_expected_results { std::unique_ptr<F[]>(new F[NUM_FLOATING_POINT_FUNCTION_TEST_LINES]) };

	cuda::launch(
		kernels::try_out_floating_point_math_functions<F>,
		launch_config,
		device_side_results.get(), device_side_expected_results.get());

	cuda::memory::copy(host_side_results.get(), device_side_results.get(), sizeof(F) * NUM_FLOATING_POINT_FUNCTION_TEST_LINES);
	cuda::memory::copy(host_side_expected_results.get(), device_side_expected_results.get(), sizeof(F) * NUM_FLOATING_POINT_FUNCTION_TEST_LINES);

	check_results(
		NUM_FLOATING_POINT_FUNCTION_TEST_LINES,
		host_side_results.get(),
		[ expected_results = host_side_expected_results.get() ](std::size_t i) { return expected_results[i]; },
		optional<F>{0.00001});
}


} // TEST_SUITE("math")
