#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "common.cuh"
#include <kat/on_device/math.cuh>

namespace kernels {

template <typename I>
__global__ void try_out_integral_math_functions(I* results, I* __restrict expected)
{
	size_t i { 0 };
	bool print_first_indices_for_each_function { false };

	auto maybe_print = [&](const char* section_title) {
		if (print_first_indices_for_each_function) {
			printf("%-30s tests start at index  %3d\n", section_title, (int) i);
		}
	};

	results[i] = kat::strictly_between<I>( I{   0 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   1 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   4 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   5 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{   6 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{   8 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{   9 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::strictly_between<I>( I{  10 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{  11 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::strictly_between<I>( I{ 123 }, I{  5 }, I{  10 } ); expected[i++] = false;

	maybe_print("between_or_equal");
	results[i] = kat::between_or_equal<I>( I{   1 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{   4 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{   5 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   6 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   8 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{   9 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{  10 }, I{  5 }, I{  10 } ); expected[i++] = true;
	results[i] = kat::between_or_equal<I>( I{  11 }, I{  5 }, I{  10 } ); expected[i++] = false;
	results[i] = kat::between_or_equal<I>( I{ 123 }, I{  5 }, I{  10 } ); expected[i++] = false;

	maybe_print("is_power_of_2");
	results[i] = kat::is_power_of_2<I>(I{ 1}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 2}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 4}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{ 7}); expected[i++] = false;
	results[i] = kat::is_power_of_2<I>(I{32}); expected[i++] = true;
	results[i] = kat::is_power_of_2<I>(I{33}); expected[i++] = false;

	maybe_print("modular_increment");
	results[i] = kat::modular_increment<I>(I{ 0}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 1}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 0}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_increment<I>(I{ 1}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_increment<I>(I{ 2}, I{ 3}); expected[i++] = I{ 0 };
	results[i] = kat::modular_increment<I>(I{ 3}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_increment<I>(I{ 4}, I{ 3}); expected[i++] = I{ 2 };

	maybe_print("modular_decrement");
	results[i] = kat::modular_decrement<I>(I{ 0}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 1}, I{ 1}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 0}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_decrement<I>(I{ 1}, I{ 3}); expected[i++] = I{ 0 };
	results[i] = kat::modular_decrement<I>(I{ 2}, I{ 3}); expected[i++] = I{ 1 };
	results[i] = kat::modular_decrement<I>(I{ 3}, I{ 3}); expected[i++] = I{ 2 };
	results[i] = kat::modular_decrement<I>(I{ 4}, I{ 3}); expected[i++] = I{ 0 };

	maybe_print("ipow");
	results[i] = kat::ipow<I>(I{ 0 },   1 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 0 },   2 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 0 }, 100 ); expected[i++] = I{  0 };
	results[i] = kat::ipow<I>(I{ 1 },   0 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 },   1 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 },   2 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 1 }, 100 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 3 },   0 ); expected[i++] = I{  1 };
	results[i] = kat::ipow<I>(I{ 3 },   1 ); expected[i++] = I{  3 };
	results[i] = kat::ipow<I>(I{ 3 },   2 ); expected[i++] = I{  9 };
	results[i] = kat::ipow<I>(I{ 3 },   4 ); expected[i++] = I{ 81 };

	maybe_print("unsafe div_rounding_up");
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };

	maybe_print("div_rounding_up");
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::div_rounding_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{   1 };
	results[i] = kat::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( std::numeric_limits<I>::max()    , std::numeric_limits<I>::max() - 1 ); expected[i++] = I{   2 };
	results[i] = kat::div_rounding_up<I>( std::numeric_limits<I>::max() - 1, std::numeric_limits<I>::max()     ); expected[i++] = I{   1 };

	maybe_print("round_down");
	results[i] = kat::round_down<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{   1 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{ 122 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down<I>( I{ 123 }, I{ 123 } ); expected[i++] = I{ 123 };
	results[i] = kat::round_down<I>( I{ 124 }, I{ 123 } ); expected[i++] = I{ 123 };

	maybe_print("round_down_to_full_warps");
	results[i] = kat::round_down_to_full_warps<I>( I{   0 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{   1 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{   8 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  16 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  31 } ); expected[i++] = I{  0 };
	results[i] = kat::round_down_to_full_warps<I>( I{  32 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_down_to_full_warps<I>( I{  33 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_down_to_full_warps<I>( I{ 125 } ); expected[i++] = I{ 96 };

	// TODO: Consider testing rounding-up with negative dividends

	maybe_print("unsafe round_up");
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::unsafe::round_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::round_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up<I>( I{  63 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::unsafe::round_up<I>( I{  64 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::unsafe::round_up<I>( I{  65 }, I{  32 } ); expected[i++] = I{  96 };

	maybe_print("round_up");
	results[i] = kat::round_up<I>( I{   0 }, I{   1 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   0 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   0 }, I{ 123 } ); expected[i++] = I{   0 };
	results[i] = kat::round_up<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_up<I>( I{   1 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up<I>( I{  63 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::round_up<I>( I{  64 }, I{  64 } ); expected[i++] = I{  64 };
	results[i] = kat::round_up<I>( I{  65 }, I{  32 } ); expected[i++] = I{  96 };
	results[i] = kat::round_up<I>( std::numeric_limits<I>::max() - 1, std::numeric_limits<I>::max() ); expected[i++] = I{ std::numeric_limits<I>::max() };

	maybe_print("round_down_to_power_of_2");
	results[i] = kat::round_down_to_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   3 }, I{   1 } ); expected[i++] = I{   3 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   4 }, I{   1 } ); expected[i++] = I{   4 };
	results[i] = kat::round_down_to_power_of_2<I>( I{ 123 }, I{   1 } ); expected[i++] = I{ 123 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = I{   0 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   3 }, I{   2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_down_to_power_of_2<I>( I{   4 }, I{   2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_down_to_power_of_2<I>( I{ 123 }, I{   2 } ); expected[i++] = I{ 122 };

	maybe_print("round_up_to_power_of_2");
	results[i] = kat::round_up_to_power_of_2<I>( I{  1 }, I{  1 } ); expected[i++] = I{   1 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  2 }, I{  1 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  3 }, I{  1 } ); expected[i++] = I{   3 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  4 }, I{  1 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{ 23 }, I{  1 } ); expected[i++] = I{  23 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  1 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  2 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  3 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{  4 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::round_up_to_power_of_2<I>( I{ 63 }, I{  2 } ); expected[i++] = I{  64 };

	maybe_print("unsafe round_up_to_power_of_2");
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  1 }, I{  1 } ); expected[i++] = I{   1 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  2 }, I{  1 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  3 }, I{  1 } ); expected[i++] = I{   3 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  4 }, I{  1 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{ 23 }, I{  1 } ); expected[i++] = I{  23 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  1 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  2 }, I{  2 } ); expected[i++] = I{   2 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  3 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{  4 }, I{  2 } ); expected[i++] = I{   4 };
	results[i] = kat::unsafe::round_up_to_power_of_2<I>( I{ 63 }, I{  2 } ); expected[i++] = I{  64 };

	maybe_print("round_up_to_full_warps");
	results[i] = kat::round_up_to_full_warps<I>( I{   0 } ); expected[i++] = I{  0 };
	results[i] = kat::round_up_to_full_warps<I>( I{   1 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{   8 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  16 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  31 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  32 } ); expected[i++] = I{ 32 };
	results[i] = kat::round_up_to_full_warps<I>( I{  33 } ); expected[i++] = I{ 64 };
	results[i] = kat::round_up_to_full_warps<I>( I{  63 } ); expected[i++] = I{ 64 };

	maybe_print("gcd");
	results[i] = kat::gcd<I>( I{   1 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   2 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   1 }, I{   2 } ); expected[i++] = I{  1 };
	results[i] = kat::gcd<I>( I{   2 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::gcd<I>( I{   8 }, I{   4 } ); expected[i++] = I{  4 };
	results[i] = kat::gcd<I>( I{   4 }, I{   8 } ); expected[i++] = I{  4 };
	results[i] = kat::gcd<I>( I{  10 }, I{   6 } ); expected[i++] = I{  2 };
	results[i] = kat::gcd<I>( I{ 120 }, I{  70 } ); expected[i++] = I{ 10 };
	results[i] = kat::gcd<I>( I{  70 }, I{ 120 } ); expected[i++] = I{ 10 };
	results[i] = kat::gcd<I>( I{  97 }, I{ 120 } ); expected[i++] = I{  1 };

	maybe_print("lcm");
	results[i] = kat::lcm<I>( I{   1 }, I{   1 } ); expected[i++] = I{  1 };
	results[i] = kat::lcm<I>( I{   2 }, I{   1 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   1 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   2 }, I{   2 } ); expected[i++] = I{  2 };
	results[i] = kat::lcm<I>( I{   5 }, I{   3 } ); expected[i++] = I{ 15 };
	results[i] = kat::lcm<I>( I{   8 }, I{   4 } ); expected[i++] = I{  8 };
	results[i] = kat::lcm<I>( I{   4 }, I{   8 } ); expected[i++] = I{  8 };
	results[i] = kat::lcm<I>( I{  10 }, I{   6 } ); expected[i++] = I{ 30 };

	maybe_print("is_even");
	results[i] = kat::is_even<I>( I{   0 } ); expected[i++] = true;
	results[i] = kat::is_even<I>( I{   1 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{   2 } ); expected[i++] = true;
	results[i] = kat::is_even<I>( I{   3 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{ 123 } ); expected[i++] = false;
	results[i] = kat::is_even<I>( I{ 124 } ); expected[i++] = true;

	maybe_print("is_odd");
	results[i] = kat::is_odd<I>( I{   0 } ); expected[i++] = false;
	results[i] = kat::is_odd<I>( I{   1 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{   2 } ); expected[i++] = false;
	results[i] = kat::is_odd<I>( I{   3 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{ 123 } ); expected[i++] = true;
	results[i] = kat::is_odd<I>( I{ 124 } ); expected[i++] = false;

	maybe_print("log2");
	results[i] = kat::log2<I>( I{   1 } ); expected[i++] = 0;
	results[i] = kat::log2<I>( I{   2 } ); expected[i++] = 1;
	results[i] = kat::log2<I>( I{   3 } ); expected[i++] = 1;
	results[i] = kat::log2<I>( I{   4 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   6 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   7 } ); expected[i++] = 2;
	results[i] = kat::log2<I>( I{   8 } ); expected[i++] = 3;
	results[i] = kat::log2<I>( I{ 127 } ); expected[i++] = 6;

//	We don't have a goot integer sqrt() implementation to offer here. Perhaps
//	we could offer something based on casting to float?
//
//	results[i] = kat::sqrt<I>( I{   0 } ); expected[i++] =  0;
//	results[i] = kat::sqrt<I>( I{   1 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   2 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   3 } ); expected[i++] =  1;
//	results[i] = kat::sqrt<I>( I{   4 } ); expected[i++] =  2;
//	results[i] = kat::sqrt<I>( I{   5 } ); expected[i++] =  2;
//	results[i] = kat::sqrt<I>( I{   9 } ); expected[i++] =  3;
//	results[i] = kat::sqrt<I>( I{  10 } ); expected[i++] =  3;
//	results[i] = kat::sqrt<I>( I{ 127 } ); expected[i++] = 11;

	maybe_print("div_by_power_of_2");
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I {  1 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I {  1 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I {  1 }); expected[i++] = I{ 111 };
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I {  2 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I {  2 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   2 }, I {  2 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{   3 }, I {  2 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{   4 }, I {  2 }); expected[i++] = I{   2 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I {  2 }); expected[i++] = I{  55 };
	results[i] = kat::div_by_power_of_2<I>( I{   0 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{   1 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{  15 }, I { 16 }); expected[i++] = I{   0 };
	results[i] = kat::div_by_power_of_2<I>( I{  16 }, I { 16 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{  17 }, I { 16 }); expected[i++] = I{   1 };
	results[i] = kat::div_by_power_of_2<I>( I{  32 }, I { 16 }); expected[i++] = I{   2 };
	results[i] = kat::div_by_power_of_2<I>( I{ 111 }, I { 16 }); expected[i++] = I{   6 };

	maybe_print("divides");
	results[i] = kat::divides<I>( I{   1 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   3 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   3 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   1 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{   3 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   4 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   6 }, I{   9 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   9 }, I{   6 } ); expected[i++] = false;
	results[i] = kat::divides<I>( I{   4 }, I{  24 } ); expected[i++] = true;
	results[i] = kat::divides<I>( I{  24 }, I{   4 } ); expected[i++] = false;

	maybe_print("is_divisible_by");
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   0 }, I{   3 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   1 }, I{   3 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   3 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   2 }, I{   4 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   9 }, I{   6 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{   6 }, I{   9 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by<I>( I{  24 }, I{   4 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by<I>( I{   4 }, I{  24 } ); expected[i++] = false;

	maybe_print("is_divisible_by_power_of_2");
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   0 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   0 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{   2 }, I{   4 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  24 }, I{   4 } ); expected[i++] = true;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  72 }, I{  16 } ); expected[i++] = false;
	results[i] = kat::is_divisible_by_power_of_2<I>( I{  64 }, I{  16 } ); expected[i++] = true;

	maybe_print("power_of_2_divides");
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   0 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   1 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   1 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{   1 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   2 }, I{   2 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{   4 }, I{   2 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{   4 }, I{  24 } ); expected[i++] = true;
	results[i] = kat::power_of_2_divides<I>( I{  16 }, I{  72 } ); expected[i++] = false;
	results[i] = kat::power_of_2_divides<I>( I{  16 }, I{  64 } ); expected[i++] = true;

	maybe_print("log2_of_power_of_2");
	results[i] = kat::log2_of_power_of_2<I>( I{  1 } ); expected[i++] = I{ 0 };
	results[i] = kat::log2_of_power_of_2<I>( I{  2 } ); expected[i++] = I{ 1 };
	results[i] = kat::log2_of_power_of_2<I>( I{  4 } ); expected[i++] = I{ 2 };
	results[i] = kat::log2_of_power_of_2<I>( I{  8 } ); expected[i++] = I{ 3 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 16 } ); expected[i++] = I{ 4 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 32 } ); expected[i++] = I{ 5 };
	results[i] = kat::log2_of_power_of_2<I>( I{ 64 } ); expected[i++] = I{ 6 };

	maybe_print("modulo_power_of_2");
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   1 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   2 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   2 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   0 }, I{   4 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   1 }, I{   4 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{   2 }, I{   4 } ); expected[i++] = I{ 2 };
	results[i] = kat::modulo_power_of_2<I>( I{   3 }, I{   4 } ); expected[i++] = I{ 3 };
	results[i] = kat::modulo_power_of_2<I>( I{   4 }, I{   4 } ); expected[i++] = I{ 0 };
	results[i] = kat::modulo_power_of_2<I>( I{   5 }, I{   4 } ); expected[i++] = I{ 1 };
	results[i] = kat::modulo_power_of_2<I>( I{  63 }, I{   4 } ); expected[i++] = I{ 3 };

#define NUM_TEST_LINES 268

}

} // namespace kernels

TEST_SUITE("math") {

TEST_CASE_TEMPLATE("run-time on-device", I, INTEGER_TYPES)
{
	cuda::device_t device { cuda::device::current::get() };
	auto block_size { 1 };
	auto num_grid_blocks { 1 };
	auto launch_config { cuda::make_launch_config(block_size, num_grid_blocks) };
	auto device_side_results { cuda::memory::device::make_unique<I[]>(device, NUM_TEST_LINES) };
	auto device_side_expected_results { cuda::memory::device::make_unique<I[]>(device, NUM_TEST_LINES) };
	auto host_side_results { std::unique_ptr<I[]>(new I[NUM_TEST_LINES]) };
	auto host_side_expected_results { std::unique_ptr<I[]>(new I[NUM_TEST_LINES]) };

	cuda::launch(
		kernels::try_out_integral_math_functions<I>,
		launch_config,
		device_side_results.get(), device_side_expected_results.get());

	cuda::memory::copy(host_side_results.get(), device_side_results.get(), sizeof(I) * NUM_TEST_LINES);
	cuda::memory::copy(host_side_expected_results.get(), device_side_expected_results.get(), sizeof(I) * NUM_TEST_LINES);

	for(auto i { 0 }; i < NUM_TEST_LINES; i++) {
		CHECK(host_side_results.get()[i] == host_side_expected_results.get()[i]);
		if (host_side_results.get()[i] != host_side_expected_results.get()[i]) {
			MESSAGE("index of failure was: " << i);
		}
	}
}

} // TEST_SUITE("math")
